#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define MAX_SIZE 1000

__global__ void kernel_add(int* v1, int* v2, int* vRes) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index<MAX_SIZE) {
        vRes[index] = v1[index] + v2[index];
    }
}

int main() {
    float elapsed_time;
    size_t size = sizeof(int)*MAX_SIZE;

    // Allocate CPU Memory
    int* arr1 = (int*)malloc(size);
    int* arr2 = (int*)malloc(size);
    int* arrRes = (int*)malloc(size);

    // Allocate GPU Memory
    int *Darr1,*Darr2,*DarrRes;
    hipMalloc(&Darr1,size);
    hipMalloc(&Darr2,size);
    hipMalloc(&DarrRes,size);
    
    if (arr1 == NULL || arr2 == NULL || arrRes == NULL || Darr1 == NULL || Darr2 == NULL || DarrRes == NULL) {
        fprintf(stderr, "Failed to allocate memory on host\n");
        return 1;
    }

    for(int index=0;index<MAX_SIZE;index++) {
        arr1[index] = index;
        arr2[index] = index * MAX_SIZE;
    }

    // Copy data to GPU
    hipMemcpy(Darr1,arr1,size,hipMemcpyHostToDevice);
    hipMemcpy(Darr2,arr2,size,hipMemcpyHostToDevice);
    
    // START events
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // START Recording time
    hipEventRecord(start,0);
    dim3 threads_per_block(256);
    dim3 blocks_per_grid((MAX_SIZE + threads_per_block.x - 1) / threads_per_block.x);
    kernel_add<<<blocks_per_grid,threads_per_block>>>(Darr1,Darr2,DarrRes);
    // STOP Recording time
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipMemcpy(arrRes,DarrRes,size,hipMemcpyDeviceToHost);
    hipEventElapsedTime(&elapsed_time,start,stop);
    
    printf("Parallel Execution Time: %f",elapsed_time);

    // Free Memory
    free(arr1);
    free(arr2);
    free(arrRes);
    hipFree(Darr1);
    hipFree(Darr2);
    hipFree(DarrRes);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("\n");
    return EXIT_SUCCESS;
}