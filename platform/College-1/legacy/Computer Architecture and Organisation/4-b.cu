#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define MAX_SIZE 1000

__global__ void kernel_add(int* v1, int* v2, int* vRes) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index<MAX_SIZE) {
        vRes[index] = v1[index] + v2[index];
    }
}

void serial_add(int* v1, int* v2, int* vRes) {
    for(int index=0;index<MAX_SIZE;index++) {
        vRes[index] = v1[index]+v2[index];
    }
}

int main() {
    float elapsed_time_parallel, elapsed_time_serial;
    size_t size = sizeof(int)*MAX_SIZE;

    // Allocate CPU Memory
    int* arr1 = (int*)malloc(size);
    int* arr2 = (int*)malloc(size);
    int* arrRes = (int*)malloc(size);

    // Allocate GPU Memory
    int *Darr1,*Darr2,*DarrRes;
    hipMalloc(&Darr1,size);
    hipMalloc(&Darr2,size);
    hipMalloc(&DarrRes,size);
    
    if (arr1 == NULL || arr2 == NULL || arrRes == NULL || Darr1 == NULL || Darr2 == NULL || DarrRes == NULL) {
        fprintf(stderr, "Failed to allocate memory on host\n");
        return 1;
    }

    for(int index=0;index<MAX_SIZE;index++) {
        arr1[index] = index;
        arr2[index] = index * MAX_SIZE;
    }
    // Copy data to GPU
    hipMemcpy(Darr1,arr1,size,hipMemcpyHostToDevice);
    hipMemcpy(Darr2,arr2,size,hipMemcpyHostToDevice);
    // Parallel Execution
    // START Recording event
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);
    dim3 threads_per_block(256);
    dim3 blocks_per_grid((MAX_SIZE + threads_per_block.x - 1) / threads_per_block.x);
    kernel_add<<<blocks_per_grid,threads_per_block>>>(Darr1,Darr2,DarrRes);
    // STOP Recording event
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipMemcpy(arrRes,DarrRes,size,hipMemcpyDeviceToHost);
    hipEventElapsedTime(&elapsed_time_parallel,start,stop);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Serial Execution
    // START Recording event
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start,0);
    serial_add(arr1,arr2,arrRes);
    // STOP Recording event
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_serial,start,stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    printf("Parallel Execution Time: %f",elapsed_time_parallel);
    printf("\nSerial Execution Time: %f",elapsed_time_serial);
    printf("\nDifference: %f",elapsed_time_parallel-elapsed_time_serial);

    // Free Memory
    free(arr1);
    free(arr2);
    free(arrRes);
    hipFree(Darr1);
    hipFree(Darr2);
    hipFree(DarrRes);

    printf("\n");
    return EXIT_SUCCESS;
}